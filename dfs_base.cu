#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"

#define ZERO 0

extern float toBW(int bytes, float sec);

__global__ void
zero_edge_weights(int M, int* edge_weights) {
  // compute overall index from position of thread in current block,
  // and given the block we are in
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index < M) {
    edge_weights[index] = 0;
  }
}

__global__ void
setup_zeta_leaves(int N, int* zeta, bool* leaves, bool* q_queue, int* child_counter) {
  // compute overall index from position of thread in current block,
  // and given the block we are in
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index <= N) {
    if (leaves[index]) {
        zeta[index] = 1;
        q_queue[index] = true;
    }
    child_counter[index] = 0;
  }
}

__global__ void
propagate_zeta(int N, int* zeta, int* edge_weights, bool* q_queue, bool* c_queue, int* offsets, 
    int* neighbours, int* p_offsets, int* parents, int* child_to_parent) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index <= N) {
        if (q_queue[index]) {
            int child_offset = p_offsets[index];
            int num_parents = p_offsets[index + 1] - child_offset;
            // TODO : Parallelizing this for loop might help
            for (int i = 0; i < num_parents; i++) {
                int j = child_to_parent[child_offset + i];
                edge_weights[j] = zeta[index];
            }
        }
    }
}

__global__ void
calculate_parent_zeta(int N, int* zeta, int* edge_weights, bool* q_queue, bool* c_queue, int* offsets, 
    int* neighbours, int* p_offsets, int* parents, int* child_to_parent) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index <= N && zeta[index] == 0) {
        int n_offset = offsets[index];
        int n_children = offsets[index + 1] - n_offset;

        bool flag = true;

        for (int i = 0; i < n_children; i++) {
            if (edge_weights[n_offset + i] == 0) {
                flag = false;
                break;
            }
        }

        if(flag) {
            int prefix_sum = 1;
            for (int i = 0; i < n_children; i++) {
                int temp = edge_weights[n_offset + i];
                edge_weights[n_offset + i] = prefix_sum;
                prefix_sum += temp;
            }
            zeta[index] = prefix_sum;
            c_queue[index] = true;
        }
    }
}

__global__ void
exchange_c_q(int N, bool* q_queue, bool* c_queue) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index <= N) {
        q_queue[index] = c_queue[index];
        c_queue[index] = 0;
    }
}

void
DfsCuda(int N, int M, int* offsets, int* neighbours, bool* leaves, int* p_offsets, 
    int* parents, int* child_to_parent, int** results, int* zeta) {

    int totalBytes = sizeof(int) * (6 * N + 3 * M + 8) + sizeof(bool) * (N + 1);

    // start timing
    double startTime = CycleTimer::currentSeconds();

    // compute number of blocks and threads per block
    const int threadsPerBlock = 256;
    const int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;
    const int blocks_edges = (M + threadsPerBlock - 1) / threadsPerBlock;
    int nnodes = N;
    int nedges = M;

    int* device_offsets;
    int* device_neighbours;
    bool* device_leaves;
    int* device_p_offsets;
    int* device_parents;
    int** device_results;
    int* device_zeta;
    int* device_child_to_parent;

    int* device_edge_weights;
    int* child_counter;
    bool* c_queue;
    bool* q_queue;
    //
    // allocate device memory buffers on the GPU using hipMalloc
    //
    hipMalloc(&device_offsets, (nnodes+2) * sizeof(int));
    hipMalloc(&device_neighbours, nedges * sizeof(int));
    hipMalloc(&device_leaves, (nnodes+1) * sizeof(bool));
    hipMalloc(&device_p_offsets, (nnodes + 2) * sizeof(int));
    hipMalloc(&device_parents, nedges * sizeof(int));
    hipMalloc(&device_results, 3 * (nnodes + 1) * sizeof(int));
    hipMalloc(&device_zeta, (nnodes + 1) * sizeof(int));
    hipMalloc(&device_child_to_parent, (nedges) * sizeof(int));

    hipMalloc(&device_edge_weights, nedges * sizeof(int));
    hipMalloc(&c_queue, (nnodes + 1) * sizeof(bool));
    hipMalloc(&q_queue, (nnodes + 1) * sizeof(bool));
    hipMalloc(&child_counter, (nnodes + 1)*sizeof(int));

    hipMemset(c_queue, false, (nnodes + 1) * sizeof(bool));
    hipMemset(q_queue, false, (nnodes + 1) * sizeof(bool));

    //
    // copy input arrays to the GPU using hipMemcpy
    //
    hipMemcpy(device_offsets, offsets, (nnodes+2) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_neighbours, neighbours, nedges * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_leaves, leaves, (nnodes + 1) * sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(device_p_offsets, p_offsets, (nnodes + 2) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_parents, parents, nedges * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_results, results, 3 * (nnodes + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_zeta, zeta, (nnodes + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_child_to_parent, child_to_parent, (nedges) * sizeof(int), hipMemcpyHostToDevice);


    // run kernel
    double startTime2 = CycleTimer::currentSeconds();

    // Run DFS on GPU

    // Phase 1 (Calculate zeta of nodes)

    // setup the zeta's for leaves and initialize q with the leaves
    zero_edge_weights<<<blocks_edges, threadsPerBlock>>>(nedges, device_edge_weights);
    hipDeviceSynchronize();
    setup_zeta_leaves<<<blocks, threadsPerBlock>>>(nnodes, device_zeta, device_leaves, q_queue, child_counter);
    hipDeviceSynchronize();
    int* edge_weights = (int *) malloc(sizeof(int) * (nedges));

    // calculate edge weights
    while(true) {
        int zeta_of_zero = 0;

        propagate_zeta<<<blocks, threadsPerBlock>>>(nnodes, device_zeta, device_edge_weights, q_queue, 
            c_queue, device_offsets, device_neighbours, device_p_offsets, device_parents, device_child_to_parent);
        hipDeviceSynchronize();

        calculate_parent_zeta<<<blocks, threadsPerBlock>>>(nnodes, device_zeta, device_edge_weights, q_queue, 
            c_queue, device_offsets, device_neighbours, device_p_offsets, device_parents, device_child_to_parent);
        hipDeviceSynchronize();

        exchange_c_q<<<blocks, threadsPerBlock>>>(nnodes, q_queue, c_queue);
        hipDeviceSynchronize();

        hipMemcpy(&zeta_of_zero, device_zeta, 1 * sizeof(int), hipMemcpyDeviceToHost);
        if (zeta_of_zero) {
            break;
        }
    }

    hipDeviceSynchronize();

    double endTime2 = CycleTimer::currentSeconds();

    //
    // copy result from GPU using hipMemcpy
    //
    hipMemcpy(zeta, device_zeta, (nnodes+1) * sizeof(int), hipMemcpyDeviceToHost);

    hipMemcpy(edge_weights, device_edge_weights, nedges * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i<=nnodes; i++) {
        int offset = offsets[i];
        for (int j = 0; j < (offsets[i+1] - offsets[i]); j++) {
            int child = neighbours[offset + j];
            std::cout << i << " - " << child << " and edge weight " << edge_weights[offset + j] << "\n";
        }
    }
    free(edge_weights);

    // end timing after result has been copied back into host memory
    double endTime = CycleTimer::currentSeconds();

    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
    }

    double overallDuration = endTime - startTime;
    double overallDuration2 = endTime2 - startTime2;
    printf("Kernel Running Time: %.3f ms\n", 1000.f * overallDuration2);
    printf("Overall: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * overallDuration, toBW(totalBytes, overallDuration));

    // free memory buffers on the GPU
    hipFree(device_offsets);
    hipFree(device_neighbours);
    hipFree(device_leaves);
    hipFree(device_p_offsets);
    hipFree(device_parents);
    hipFree(device_results);
    hipFree(device_zeta);
    hipFree(device_child_to_parent);

    hipFree(device_edge_weights);
    hipFree(child_counter);
    hipFree(c_queue);
    hipFree(q_queue);
}

void
printCudaInfo() {

    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
