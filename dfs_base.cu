#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"

#define ZERO 0

extern float toBW(int bytes, float sec);

__global__ void
zero_edge_weights(int M, int* edge_weights) {
  // compute overall index from position of thread in current block,
  // and given the block we are in
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index < M) {
    edge_weights[index] = 0;
  }
}

__global__ void
setup_zeta_leaves(int N, int* zeta, bool* leaves, bool* q_queue) {
  // compute overall index from position of thread in current block,
  // and given the block we are in
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index <= N) {
    if (leaves[index]) {
        zeta[index] = 1;
        q_queue[index] = true;
    }
  }
}

__global__ void
calculate_edge_weights(int N, int* zeta, int* edge_weights, bool* q_queue, bool* c_queue, 
    int* offsets, int* neighbours, int* p_offsets, int* parents, int* child_to_parent) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index <= N) {
        if (q_queue[index]) {
            int child_offset = p_offsets[index];
            int num_parents = p_offsets[index + 1] - child_offset;
            for (int i = 0; i < num_parents; i++) {
                int index = child_to_parent[child_offset + i];
                edge_weights[index] = zeta[index];
            }
        }
        else if (zeta[index] == 0) {
            bool flag = true;
            int n_offset = offsets[index];
            int n_children = offsets[index + 1] - n_offset;
            for (int i = 0; i < n_children; i++) {
                if (edge_weights[n_offset + i] == 0) {
                    flag = false;
                    break;
                }
            }
            if (flag) {
                int prefix_sum = 1;
                for (int i = 0; i < n_children; i++) {
                    int temp = edge_weights[n_offset + i];
                    edge_weights[n_offset + i] = prefix_sum;
                    prefix_sum += temp;
                }
                zeta[index] = prefix_sum;
                c_queue[index] = true;
            }
        }
    }
}

void
DfsCuda(int N, int M, int* offsets, int* neighbours, bool* leaves, int* p_offsets, 
    int* parents, int* child_to_parent, int** results, int* zeta) {

    int totalBytes = sizeof(int) * (6 * N + 3 * M + 8) + sizeof(bool) * (N + 1);

    // start timing
    double startTime = CycleTimer::currentSeconds();

    // compute number of blocks and threads per block
    const int threadsPerBlock = 256;
    const int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;
    const int blocks_edges = (M + threadsPerBlock - 1) / threadsPerBlock;
    int nnodes = N;
    int nedges = M;

    int* device_offsets;
    int* device_neighbours;
    bool* device_leaves;
    int* device_p_offsets;
    int* device_parents;
    int** device_results;
    int* device_zeta;
    int* device_child_to_parent;

    int* device_edge_weights;
    bool* c_queue;
    bool* q_queue;
    //
    // allocate device memory buffers on the GPU using hipMalloc
    //
    hipMalloc(&device_offsets, (nnodes+2) * sizeof(int));
    hipMalloc(&device_neighbours, nedges * sizeof(int));
    hipMalloc(&device_leaves, (nnodes+1) * sizeof(bool));
    hipMalloc(&device_p_offsets, (nnodes + 2) * sizeof(int));
    hipMalloc(&device_parents, nedges * sizeof(int));
    hipMalloc(&device_results, 3 * (nnodes + 1) * sizeof(int));
    hipMalloc(&device_zeta, (nnodes + 1) * sizeof(int));
    hipMalloc(&device_child_to_parent, (nedges) * sizeof(int));

    hipMalloc(&device_edge_weights, nedges * sizeof(int));
    hipMalloc(&c_queue, (nnodes + 1) * sizeof(bool));
    hipMalloc(&q_queue, (nnodes + 1) * sizeof(bool));

    hipMemset(c_queue, false, (nnodes + 1) * sizeof(bool));
    hipMemset(q_queue, false, (nnodes + 1) * sizeof(bool));

    //
    // copy input arrays to the GPU using hipMemcpy
    //
    hipMemcpy(device_offsets, offsets, (nnodes+2) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_neighbours, neighbours, nedges * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_leaves, leaves, (nnodes + 1) * sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(device_p_offsets, p_offsets, (nnodes + 2) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_parents, parents, nedges * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_results, results, 3 * (nnodes + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_zeta, zeta, (nnodes + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_child_to_parent, child_to_parent, (nedges) * sizeof(int), hipMemcpyHostToDevice);


    // run kernel
    double startTime2 = CycleTimer::currentSeconds();

    // Run DFS on GPU

    // Phase 1 (Calculate zeta of nodes)

    // setup the zeta's for leaves and initialize q with the leaves
    zero_edge_weights<<<blocks_edges, threadsPerBlock>>>(nedges, device_edge_weights);
    hipDeviceSynchronize();
    setup_zeta_leaves<<<blocks, threadsPerBlock>>>(nnodes, device_zeta, device_leaves, q_queue);
    hipDeviceSynchronize();

    // calculate edge weights
    calculate_edge_weights<<<blocks, threadsPerBlock>>>(nnodes, device_zeta, device_edge_weights, q_queue, 
        c_queue, device_offsets, device_neighbours, device_p_offsets, device_parents, device_child_to_parent);
    hipDeviceSynchronize();

    double endTime2 = CycleTimer::currentSeconds();

    //
    // copy result from GPU using hipMemcpy
    //
    hipMemcpy(zeta, device_zeta, (nnodes+1) * sizeof(int), hipMemcpyDeviceToHost);

    // end timing after result has been copied back into host memory
    double endTime = CycleTimer::currentSeconds();

    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
    }

    double overallDuration = endTime - startTime;
    double overallDuration2 = endTime2 - startTime2;
    printf("Kernel Running Time: %.3f ms\n", 1000.f * overallDuration2);
    printf("Overall: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * overallDuration, toBW(totalBytes, overallDuration));

    // free memory buffers on the GPU
    hipFree(device_offsets);
    hipFree(device_neighbours);
    hipFree(device_leaves);
    hipFree(device_p_offsets);
    hipFree(device_parents);
    hipFree(device_results);
    hipFree(device_zeta);
    hipFree(device_child_to_parent);

    hipFree(device_edge_weights);
    hipFree(c_queue);
    hipFree(q_queue);
}

void
printCudaInfo() {

    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
