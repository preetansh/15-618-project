#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"

#define ZERO 0

extern float toBW(int bytes, float sec);

__global__ void
zero_edge_weights(int M, int* edge_weights) {
  // compute overall index from position of thread in current block,
  // and given the block we are in
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index < M) {
    edge_weights[index] = 0;
  }
}

__global__ void
setup_zeta_leaves(int N, int* zeta, bool* leaves, bool* q_queue) {
  // compute overall index from position of thread in current block,
  // and given the block we are in
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index <= N) {
    if (leaves[index]) {
        zeta[index] = 1;
        q_queue[index] = true;
    }
  }
}

__global__ void
propagate_zeta(int N, int* zeta, int* edge_weights, bool* q_queue, bool* c_queue, int* offsets, 
    int* neighbours, int* p_offsets, int* parents, int* child_to_parent) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index <= N) {
        if (q_queue[index]) {
            // printf("propagating zeta for %d\n", index);
            int child_offset = p_offsets[index];
            int num_parents = p_offsets[index + 1] - child_offset;
            // TODO : Parallelizing this for loop might help
            for (int i = 0; i < num_parents; i++) {
                int j = child_to_parent[child_offset + i];
                edge_weights[j] = zeta[index];
            }
        }
    }
}

__global__ void
calculate_parent_zeta(int N, int* zeta, int* edge_weights, bool* q_queue, bool* c_queue, int* offsets, 
    int* neighbours, int* p_offsets, int* parents, int* child_to_parent) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index <= N && zeta[index] == 0) {
        int n_offset = offsets[index];
        int n_children = offsets[index + 1] - n_offset;

        bool flag = true;

        for (int i = 0; i < n_children; i++) {
            if (edge_weights[n_offset + i] == 0) {
                flag = false;
                break;
            }
        }

        if(flag) {
            int prefix_sum = 1;
            for (int i = 0; i < n_children; i++) {
                int temp = edge_weights[n_offset + i];
                edge_weights[n_offset + i] = prefix_sum;
                prefix_sum += temp;
            }
            zeta[index] = prefix_sum;
            c_queue[index] = true;
        }
    }
}

__global__ void
exchange_c_q(int N, bool* q_queue, bool* c_queue) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index <= N) {
        q_queue[index] = c_queue[index];
        c_queue[index] = 0;
    }
}

__global__ void
setup_phase_2(int N, int* cost, bool* q_queue, bool* c_queue) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index <= N) {
        if(index == 0) {
            cost[index] = 0;
            q_queue[index] = true;
        }
        else {
            cost[index] = INT_MAX; // TODO: Change this value
            q_queue[index] = false;
        }
        c_queue[index] = false;
    }
}

__global__ void
calculate_cost(int nnodes, int* cost, bool *q_queue, bool *c_queue, int* offsets, 
    int* neighbours, int* edge_weights, int* results, bool* explored) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index <= nnodes) {
        if (q_queue[index]) {
            // printf("Cost index %d\n", index);
            int c_offset = offsets[index];
            int num_children = offsets[index + 1] - c_offset;
            for (int i = 0; i < num_children; i++) {
                int child = neighbours[i + c_offset];
                int new_cost = cost[index] + edge_weights[i + c_offset];
                edge_weights[i + c_offset] = 0;
                while (atomicMin(&cost[child], new_cost) > new_cost) {} // TODO : Ask Preetansh
                int local_cost = cost[child];
                if (local_cost == new_cost) {
                    results[child] = index;
                }
            }
            explored[index] = true;
        }
    }
}

__global__ void
construct_c_queue(int nnodes, bool *c_queue, int *p_offsets, int* parents, bool* explored) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index <= nnodes && (explored[index] == false)) {
        int p_start = p_offsets[index];
        int num_parents = p_offsets[index + 1] - p_start;
        bool flag = true;
        for (int i = 0; i < num_parents; i++) {
            if (explored[parents[i+p_start]] == false) {
                flag = false;
                break;
            } 
        }
        if (flag) {
            // printf("C_queue entering : %d\n", index);
            c_queue[index] = true;
        }
    }
}

__global__ void
check_all_explored(int nnodes, bool* all_explored, bool* explored) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index <= nnodes) {
        if (explored[index] == false) {
            *all_explored = false;
        }
    }
}

void
DfsCuda(int N, int M, int* offsets, int* neighbours, bool* leaves, int* p_offsets, 
    int* parents, int* child_to_parent, int** results, int* zeta) {

    int totalBytes = sizeof(int) * (6 * N + 3 * M + 8) + sizeof(bool) * (N + 1);

    // start timing
    double startTime = CycleTimer::currentSeconds();

    // compute number of blocks and threads per block
    const int threadsPerBlock = 256;
    const int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;
    const int blocks_edges = (M + threadsPerBlock - 1) / threadsPerBlock;
    int nnodes = N;
    int nedges = M;

    int* device_offsets;
    int* device_neighbours;
    bool* device_leaves;
    int* device_p_offsets;
    int* device_parents;
    int* device_discovery;
    int* device_result_parents;
    int* device_finish;
    int* device_zeta;
    int* device_child_to_parent;

    int* device_edge_weights;
    int* device_cost;
    bool* device_c_queue;
    bool* device_q_queue;
    bool* device_explored_2;
    //
    // allocate device memory buffers on the GPU using hipMalloc
    //
    hipMalloc(&device_offsets, (nnodes+2) * sizeof(int));
    hipMalloc(&device_neighbours, nedges * sizeof(int));
    hipMalloc(&device_leaves, (nnodes+1) * sizeof(bool));
    hipMalloc(&device_p_offsets, (nnodes + 2) * sizeof(int));
    hipMalloc(&device_parents, nedges * sizeof(int));
    hipMalloc(&device_discovery, (nnodes + 1) * sizeof(int));
    hipMalloc(&device_result_parents, (nnodes + 1) * sizeof(int));
    hipMalloc(&device_finish, (nnodes + 1) * sizeof(int));
    hipMalloc(&device_zeta, (nnodes + 1) * sizeof(int));
    hipMalloc(&device_child_to_parent, (nedges) * sizeof(int));

    hipMalloc(&device_edge_weights, nedges * sizeof(int));
    hipMalloc(&device_c_queue, (nnodes + 1) * sizeof(bool));
    hipMalloc(&device_q_queue, (nnodes + 1) * sizeof(bool));
    hipMalloc(&device_cost, (nnodes + 1)*sizeof(int));
    hipMalloc(&device_explored_2, (nnodes + 1)*sizeof(int));

    hipMemset(device_c_queue, false, (nnodes + 1) * sizeof(bool));
    hipMemset(device_q_queue, false, (nnodes + 1) * sizeof(bool));
    hipMemset(device_explored_2, false, (nnodes + 1) * sizeof(bool));

    //
    // copy input arrays to the GPU using hipMemcpy
    //
    hipMemcpy(device_offsets, offsets, (nnodes+2) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_neighbours, neighbours, nedges * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_leaves, leaves, (nnodes + 1) * sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(device_p_offsets, p_offsets, (nnodes + 2) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_parents, parents, nedges * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_discovery, results[0], (nnodes + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_result_parents, results[1], (nnodes + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_finish, results[2], (nnodes + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_zeta, zeta, (nnodes + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_child_to_parent, child_to_parent, (nedges) * sizeof(int), hipMemcpyHostToDevice);


    // run kernel
    double startTime2 = CycleTimer::currentSeconds();

    // Run DFS on GPU
    std::cout << "Starting GPU" << "\n";
    // Phase 1 (Calculate zeta of nodes)

    // setup the zeta's for leaves and initialize q with the leaves
    zero_edge_weights<<<blocks_edges, threadsPerBlock>>>(nedges, device_edge_weights);
    hipDeviceSynchronize();
    setup_zeta_leaves<<<blocks, threadsPerBlock>>>(nnodes, device_zeta, device_leaves, device_q_queue);
    hipDeviceSynchronize();
    int* edge_weights = (int *) malloc(sizeof(int) * (nedges));

    // calculate edge weights
    while(true) {
        int zeta_of_zero = 0;

        propagate_zeta<<<blocks, threadsPerBlock>>>(nnodes, device_zeta, device_edge_weights, device_q_queue, 
            device_c_queue, device_offsets, device_neighbours, device_p_offsets, device_parents, device_child_to_parent);
        hipDeviceSynchronize();

        calculate_parent_zeta<<<blocks, threadsPerBlock>>>(nnodes, device_zeta, device_edge_weights, device_q_queue, 
            device_c_queue, device_offsets, device_neighbours, device_p_offsets, device_parents, device_child_to_parent);
        hipDeviceSynchronize();

        exchange_c_q<<<blocks, threadsPerBlock>>>(nnodes, device_q_queue, device_c_queue);
        hipDeviceSynchronize();

        // stopping condition
        hipMemcpy(&zeta_of_zero, device_zeta, 1 * sizeof(int), hipMemcpyDeviceToHost);
        if (zeta_of_zero) {
            break;
        }
    }

    std::cout << "Calculated edge weights" << "\n";

    // hipMemcpy(edge_weights, device_edge_weights, nedges * sizeof(int), hipMemcpyDeviceToHost);
    // for (int i = 0; i<=nnodes; i++) {
    //     int offset = offsets[i];
    //     for (int j = 0; j < (offsets[i+1] - offsets[i]); j++) {
    //         int child = neighbours[offset + j];
    //         std::cout << i << " - " << child << " and edge weight " << edge_weights[offset + j] << "\n";
    //     }
    // }

    // Phase 2 (Calculate cost of paths and parents)

    // setup the cost and the queues
    setup_phase_2<<<blocks, threadsPerBlock>>>(nnodes, device_cost, device_q_queue, device_c_queue);

    bool* device_all_explored;
    hipMalloc(&device_all_explored, 1 * sizeof(bool));

    while(true) {
        bool all_explored = true;              

        calculate_cost<<<blocks, threadsPerBlock>>>(nnodes, device_cost, device_q_queue, device_c_queue, 
            device_offsets, device_neighbours, device_edge_weights, device_result_parents, device_explored_2);
        hipDeviceSynchronize();

        construct_c_queue<<<blocks, threadsPerBlock>>>(nnodes, device_c_queue, device_p_offsets, 
            device_parents, device_explored_2);
        hipDeviceSynchronize();

        exchange_c_q<<<blocks, threadsPerBlock>>>(nnodes, device_q_queue, device_c_queue);
        hipDeviceSynchronize();

        hipMemcpy(device_all_explored, &all_explored, 1 * sizeof(bool), hipMemcpyHostToDevice);
        check_all_explored<<<blocks, threadsPerBlock>>>(nnodes, device_all_explored, device_explored_2);
        hipDeviceSynchronize();
        hipMemcpy(&all_explored, device_all_explored, 1 * sizeof(bool), hipMemcpyDeviceToHost);
        if (all_explored) {
            break;
        }
    } 
    hipFree(device_all_explored);

    hipDeviceSynchronize();

    double endTime2 = CycleTimer::currentSeconds();

    std::cout << "Calculated parents" << "\n";

    //
    // copy result from GPU using hipMemcpy
    //
    hipMemcpy(zeta, device_zeta, (nnodes+1) * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(results[1], device_result_parents, (nnodes+1) * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(results[0], device_cost, (nnodes+1) * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i <= nnodes; i++) {
        std::cout << "Parent of " << i << " is : " << results[1][i] << " and cost is : " << results[0][i] << "\n";
    }

    // hipMemcpy(edge_weights, device_edge_weights, nedges * sizeof(int), hipMemcpyDeviceToHost);
    // for (int i = 0; i<=nnodes; i++) {
    //     int offset = offsets[i];
    //     for (int j = 0; j < (offsets[i+1] - offsets[i]); j++) {
    //         int child = neighbours[offset + j];
    //         std::cout << i << " - " << child << " and edge weight " << edge_weights[offset + j] << "\n";
    //     }
    // }
    free(edge_weights);

    // end timing after result has been copied back into host memory
    double endTime = CycleTimer::currentSeconds();

    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
    }

    double overallDuration = endTime - startTime;
    double overallDuration2 = endTime2 - startTime2;
    printf("Kernel Running Time: %.3f ms\n", 1000.f * overallDuration2);
    printf("Overall: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * overallDuration, toBW(totalBytes, overallDuration));

    // free memory buffers on the GPU
    hipFree(device_offsets);
    hipFree(device_neighbours);
    hipFree(device_leaves);
    hipFree(device_p_offsets);
    hipFree(device_parents);
    hipFree(device_discovery);
    hipFree(device_result_parents);
    hipFree(device_finish);
    hipFree(device_zeta);
    hipFree(device_child_to_parent);

    hipFree(device_edge_weights);
    hipFree(device_cost);
    hipFree(device_c_queue);
    hipFree(device_q_queue);
    hipFree(device_explored_2);
}

void
printCudaInfo() {

    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
