#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"

#define ZERO 0

extern float toBW(int bytes, float sec);

__global__ void
zero_edge_weights(int M, long long* edge_weights) {
  // compute overall index from position of thread in current block,
  // and given the block we are in
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index < M) {
    edge_weights[index] = 0;
  }
}

__global__ void
setup_zeta_leaves(int N, long long* zeta, bool* leaves, bool* q_queue) {
  // compute overall index from position of thread in current block,
  // and given the block we are in
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index <= N) {
    if (leaves[index]) {
        zeta[index] = 1;
        q_queue[index] = true;
    }
  }
}

__global__ void
propagate_zeta(int N, long long* zeta, long long* edge_weights, bool* q_queue, bool* c_queue, int* offsets, 
    int* neighbours, int* p_offsets, int* parents, int* child_to_parent) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index <= N) {
        if (q_queue[index]) {
            // printf("propagating zeta for %d\n", index);
            int child_offset = p_offsets[index];
            int num_parents = p_offsets[index + 1] - child_offset;
            // TODO : Parallelizing this for loop might help
            for (int i = 0; i < num_parents; i++) {
                int j = child_to_parent[child_offset + i];
                edge_weights[j] = zeta[index];
            }
        }
    }
}

__global__ void
calculate_parent_zeta(int N, long long* zeta, long long* edge_weights, bool* q_queue, bool* c_queue, int* offsets, 
    int* neighbours, int* p_offsets, int* parents, int* child_to_parent) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index <= N && zeta[index] == 0) {
        int n_offset = offsets[index];
        int n_children = offsets[index + 1] - n_offset;

        bool flag = true;

        for (int i = 0; i < n_children; i++) {
            if (edge_weights[n_offset + i] == 0) {
                flag = false;
                break;
            }
        }

        if(flag) {
            long long prefix_sum = 1;
            for (int i = 0; i < n_children; i++) {
                long long temp = edge_weights[n_offset + i];
                edge_weights[n_offset + i] = prefix_sum;
                if (prefix_sum < 0) {
                    printf("Edge number %d\n", n_offset + i);
                }
                prefix_sum += temp;
            }
            zeta[index] = prefix_sum;
            c_queue[index] = true;
        }
    }
}

__global__ void
exchange_c_q(int N, bool* q_queue, bool* c_queue) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index <= N) {
        q_queue[index] = c_queue[index];
        c_queue[index] = 0;
    }
}

__global__ void
zero_child_costs(int nedges, long long* child_costs) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < nedges) {
        child_costs[index] = 0;
    }
}

__global__ void
setup_phase_2(int N, long long* cost, bool* q_queue, bool* c_queue) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index <= N) {
        if(index == 0) {
            cost[index] = 0;
            q_queue[index] = true;
        }
        else {
            cost[index] = LLONG_MAX; // TODO: Change this value
            q_queue[index] = false;
        }
        c_queue[index] = false;
    }
}


__global__ void
calculate_cost(int nnodes, long long* cost, bool *q_queue, bool *c_queue, int* offsets, 
    int* neighbours, long long* edge_weights, int* parent_to_child, long long* child_costs, bool* explored) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index <= nnodes) {
        if (q_queue[index]) {
            // printf("Cost index %d\n", index);
            int c_offset = offsets[index];
            int num_children = offsets[index + 1] - c_offset;
            for (int i = 0; i < num_children; i++) {
                long long new_cost = cost[index] + edge_weights[i + c_offset];
                int parent_to_child_index = parent_to_child[c_offset + i];
                child_costs[parent_to_child_index] = new_cost;
            }
            explored[index] = true;
        }
    }
}

__global__ void
construct_c_queue(int nnodes, bool *c_queue, int *p_offsets, int* parents,
    long long* child_costs, int* results, long long* cost, bool* explored) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index <= nnodes && (explored[index] == false)) {
        int p_start = p_offsets[index];
        int num_parents = p_offsets[index + 1] - p_start;
        bool flag = true;
        long long local_cost = LLONG_MAX;
        int local_parent = -1;
        for (int i = 0; i < num_parents; i++) {
            if (explored[parents[i+p_start]] == false) {
                flag = false;
                break;
            } 
            else {
                if (child_costs[p_start + i] < local_cost) {
                    local_cost = child_costs[p_start + i];
                    local_parent = parents[i + p_start];
                }
            }
        }
        if (flag) {
            // printf("C_queue entering : %d\n", index);
            c_queue[index] = true;
            cost[index] = local_cost;
            results[index] = local_parent;
        }
    }
}

__global__ void
check_all_explored(int nnodes, bool* all_explored, bool* explored) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index <= nnodes) {
        if (explored[index] == false) {
            *all_explored = false;
        }
    }
}

void
DfsCuda(int N, int M, int* offsets, int* neighbours, bool* leaves, int* p_offsets, 
    int* parents, int* child_to_parent, int* parent_to_child, int** results, long long* zeta) {

    int totalBytes = sizeof(int) * (6 * N + 3 * M + 8) + sizeof(bool) * (N + 1);

    // start timing
    double startTime = CycleTimer::currentSeconds();

    // compute number of blocks and threads per block
    const int threadsPerBlock = 256;
    const int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;
    const int blocks_edges = (M + threadsPerBlock - 1) / threadsPerBlock;
    int nnodes = N;
    int nedges = M;

    int* device_offsets;
    int* device_neighbours;
    bool* device_leaves;
    int* device_p_offsets;
    int* device_parents;
    int* device_discovery;
    int* device_result_parents;
    int* device_finish;
    long long* device_zeta;
    int* device_child_to_parent;
    int* device_parent_to_child;

    long long* device_edge_weights;
    long long* device_cost;
    bool* device_c_queue;
    bool* device_q_queue;
    bool* device_explored_2;
    long long* device_child_costs;
    //
    // allocate device memory buffers on the GPU using hipMalloc
    //
    hipMalloc(&device_offsets, (nnodes+2) * sizeof(int));
    hipMalloc(&device_neighbours, nedges * sizeof(int));
    hipMalloc(&device_leaves, (nnodes+1) * sizeof(bool));
    hipMalloc(&device_p_offsets, (nnodes + 2) * sizeof(int));
    hipMalloc(&device_parents, nedges * sizeof(int));
    hipMalloc(&device_discovery, (nnodes + 1) * sizeof(int));
    hipMalloc(&device_result_parents, (nnodes + 1) * sizeof(int));
    hipMalloc(&device_finish, (nnodes + 1) * sizeof(int));
    hipMalloc(&device_zeta, (nnodes + 1) * sizeof(long long));
    hipMalloc(&device_child_to_parent, (nedges) * sizeof(int));
    hipMalloc(&device_parent_to_child, (nedges) * sizeof(int));

    hipMalloc(&device_edge_weights, nedges * sizeof(long long));
    hipMalloc(&device_child_costs, nedges * sizeof(long long));
    hipMalloc(&device_c_queue, (nnodes + 1) * sizeof(bool));
    hipMalloc(&device_q_queue, (nnodes + 1) * sizeof(bool));
    hipMalloc(&device_cost, (nnodes + 1)*sizeof(long long));
    hipMalloc(&device_explored_2, (nnodes + 1)*sizeof(int));

    hipMemset(device_c_queue, false, (nnodes + 1) * sizeof(bool));
    hipMemset(device_q_queue, false, (nnodes + 1) * sizeof(bool));
    hipMemset(device_explored_2, false, (nnodes + 1) * sizeof(bool));

    //
    // copy input arrays to the GPU using hipMemcpy
    //
    hipMemcpy(device_offsets, offsets, (nnodes+2) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_neighbours, neighbours, nedges * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_leaves, leaves, (nnodes + 1) * sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(device_p_offsets, p_offsets, (nnodes + 2) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_parents, parents, nedges * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_discovery, results[0], (nnodes + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_result_parents, results[1], (nnodes + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_finish, results[2], (nnodes + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_zeta, zeta, (nnodes + 1) * sizeof(long long), hipMemcpyHostToDevice);
    hipMemcpy(device_child_to_parent, child_to_parent, (nedges) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_parent_to_child, parent_to_child, (nedges) * sizeof(int), hipMemcpyHostToDevice);


    // run kernel
    double startTime2 = CycleTimer::currentSeconds();

    // Run DFS on GPU
    std::cout << "Starting GPU" << "\n";
    // Phase 1 (Calculate zeta of nodes)

    // setup the zeta's for leaves and initialize q with the leaves
    zero_edge_weights<<<blocks_edges, threadsPerBlock>>>(nedges, device_edge_weights);
    hipDeviceSynchronize();
    setup_zeta_leaves<<<blocks, threadsPerBlock>>>(nnodes, device_zeta, device_leaves, device_q_queue);
    hipDeviceSynchronize();
    long long* edge_weights = (long long *) malloc(sizeof(long long) * (nedges));

    // calculate edge weights
    while(true) {
        long long zeta_of_zero = 0;

        propagate_zeta<<<blocks, threadsPerBlock>>>(nnodes, device_zeta, device_edge_weights, device_q_queue, 
            device_c_queue, device_offsets, device_neighbours, device_p_offsets, device_parents, device_child_to_parent);
        hipDeviceSynchronize();

        // std::cout << "__________" << "\n";

        calculate_parent_zeta<<<blocks, threadsPerBlock>>>(nnodes, device_zeta, device_edge_weights, device_q_queue, 
            device_c_queue, device_offsets, device_neighbours, device_p_offsets, device_parents, device_child_to_parent);
        hipDeviceSynchronize();

        // std::cout << "************" << "\n";

        exchange_c_q<<<blocks, threadsPerBlock>>>(nnodes, device_q_queue, device_c_queue);
        hipDeviceSynchronize();

        // stopping condition
        hipMemcpy(&zeta_of_zero, device_zeta, 1 * sizeof(long long), hipMemcpyDeviceToHost);
        if (zeta_of_zero) {
            break;
        }
    }

    std::cout << "Calculated edge weights" << "\n";

    // hipMemcpy(edge_weights, device_edge_weights, nedges * sizeof(int), hipMemcpyDeviceToHost);
    // for (int i = 0; i<=nnodes; i++) {
    //     int offset = offsets[i];
    //     for (int j = 0; j < (offsets[i+1] - offsets[i]); j++) {
    //         int child = neighbours[offset + j];
    //         if (edge_weights[offset + j] <= 0) {
    //             std::cout << i << " - " << child << " and edge weight " << edge_weights[offset + j] << "\n";
    //         }
    //     }
    // }

    // Phase 2 (Calculate cost of paths and parents)

    // setup the cost and the queues
    zero_child_costs<<<blocks_edges, threadsPerBlock>>>(nedges, device_child_costs);
    hipDeviceSynchronize();
    setup_phase_2<<<blocks, threadsPerBlock>>>(nnodes, device_cost, device_q_queue, device_c_queue);

    bool* device_all_explored;
    hipMalloc(&device_all_explored, 1 * sizeof(bool));

    while(true) {
        bool all_explored = true;             

        calculate_cost<<<blocks, threadsPerBlock>>>(nnodes, device_cost, device_q_queue, device_c_queue, device_offsets, 
            device_neighbours, device_edge_weights, device_parent_to_child, device_child_costs, device_explored_2);
        hipDeviceSynchronize();

        construct_c_queue<<<blocks, threadsPerBlock>>>(nnodes, device_c_queue, device_p_offsets, 
            device_parents, device_child_costs, device_result_parents, device_cost, device_explored_2);
        hipDeviceSynchronize();

        exchange_c_q<<<blocks, threadsPerBlock>>>(nnodes, device_q_queue, device_c_queue);
        hipDeviceSynchronize();

        hipMemcpy(device_all_explored, &all_explored, 1 * sizeof(bool), hipMemcpyHostToDevice);
        check_all_explored<<<blocks, threadsPerBlock>>>(nnodes, device_all_explored, device_explored_2);
        hipDeviceSynchronize();
        hipMemcpy(&all_explored, device_all_explored, 1 * sizeof(bool), hipMemcpyDeviceToHost);
        if (all_explored) {
            break;
        }
    } 
    hipFree(device_all_explored);

    hipDeviceSynchronize();

    double endTime2 = CycleTimer::currentSeconds();

    std::cout << "Calculated parents" << "\n";

    //
    // copy result from GPU using hipMemcpy
    //
    hipMemcpy(zeta, device_zeta, (nnodes+1) * sizeof(long long), hipMemcpyDeviceToHost);
    hipMemcpy(results[1], device_result_parents, (nnodes+1) * sizeof(int), hipMemcpyDeviceToHost);
    // hipMemcpy(results[0], device_cost, (nnodes+1) * sizeof(int), hipMemcpyDeviceToHost);

    // for (int i = 0; i <= nnodes; i++) {
    //     std::cout << "Parent of " << i << " is : " << results[1][i] << " and cost is : " << results[0][i] << "\n";
    // }

    // hipMemcpy(edge_weights, device_edge_weights, nedges * sizeof(int), hipMemcpyDeviceToHost);
    // for (int i = 0; i<=nnodes; i++) {
    //     int offset = offsets[i];
    //     for (int j = 0; j < (offsets[i+1] - offsets[i]); j++) {
    //         int child = neighbours[offset + j];
    //         std::cout << i << " - " << child << " and edge weight " << edge_weights[offset + j] << "\n";
    //     }
    // }
    free(edge_weights);

    // end timing after result has been copied back into host memory
    double endTime = CycleTimer::currentSeconds();

    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
    }

    double overallDuration = endTime - startTime;
    double overallDuration2 = endTime2 - startTime2;
    printf("Kernel Running Time: %.3f ms\n", 1000.f * overallDuration2);
    printf("Overall: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * overallDuration, toBW(totalBytes, overallDuration));

    // free memory buffers on the GPU
    hipFree(device_offsets);
    hipFree(device_neighbours);
    hipFree(device_leaves);
    hipFree(device_p_offsets);
    hipFree(device_parents);
    hipFree(device_discovery);
    hipFree(device_result_parents);
    hipFree(device_finish);
    hipFree(device_zeta);
    hipFree(device_child_to_parent);
    hipFree(device_parent_to_child);

    hipFree(device_edge_weights);
    hipFree(device_child_costs);
    hipFree(device_cost);
    hipFree(device_c_queue);
    hipFree(device_q_queue);
    hipFree(device_explored_2);
}

void
printCudaInfo() {

    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
