#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"

#define INF -1

extern float toBW(int bytes, float sec);

__global__ void
setup_levels_kernel(int N, int* levels) {
  // compute overall index from position of thread in current block,
  // and given the block we are in
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index > 1 && index <= N) {
    levels[index] = INF;
  }
}

__global__ void
bfs_baseline_kernel(int N, int curr, int* levels, int* offsets,
   int* neighbours, bool* finished ) {

    // compute overall index from position of thread in current block,
    // and given the block we are in
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= 1 && index <= N) {
      int v = index;
      if (levels[v] == curr) {
        int num_nbr = offsets[v+1] - offsets[v];
        int offset = offsets[v];
        for(int i = 0; i < num_nbr; i++) {
          int w = neighbours[offset + i];
          if (levels[w] == INF) {
            *finished = false;
            levels[w] = curr + 1;
          }
        }
      }
    }
}

void
BfsCuda(int N, int M, int* offsets, int* neighbours, int* levels) {

    int totalBytes = sizeof(int) * (2 * N + M + 3);

    // start timing
    double startTime = CycleTimer::currentSeconds();

    // compute number of blocks and threads per block
    const int threadsPerBlock = 256;
    const int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;
    int nnodes = N;
    int nedges = M;

    int* device_offsets;
    int* device_neighbours;
    int* device_levels;

    //
    // allocate device memory buffers on the GPU using hipMalloc
    //
    hipMalloc(&device_offsets, (nnodes+2) * sizeof(int));
    hipMalloc(&device_neighbours, nedges * sizeof(int));
    hipMalloc(&device_levels, (nnodes+1) * sizeof(int));


    //
    // copy input arrays to the GPU using hipMemcpy
    //
    hipMemcpy(device_offsets, offsets, (nnodes+2) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_neighbours, neighbours, nedges * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_levels, levels, (nnodes + 1) * sizeof(int), hipMemcpyHostToDevice);


    // run kernel
    double startTime2 = CycleTimer::currentSeconds();
    // setup the levels array
    setup_levels_kernel<<<blocks, threadsPerBlock>>>(nnodes, device_levels);
    hipDeviceSynchronize();

    // // run bfs_baseline_kernel
    int curr = 0;
    bool finished = true;
    do {
      finished = true;
      bfs_baseline_kernel<<<blocks, threadsPerBlock>>>(nnodes, curr++,
        device_levels, device_offsets, device_neighbours, &finished);
      hipDeviceSynchronize();
    } while(!finished);
    double endTime2 = CycleTimer::currentSeconds();

    //
    // copy result from GPU using hipMemcpy
    //
    hipMemcpy(levels, device_levels, (nnodes+1) * sizeof(int), hipMemcpyDeviceToHost);

    // end timing after result has been copied back into host memory
    double endTime = CycleTimer::currentSeconds();

    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
    }

    double overallDuration = endTime - startTime;
    double overallDuration2 = endTime2 - startTime2;
    printf("Running Time: %.3f ms\n", 1000.f * overallDuration2);
    printf("Overall: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * overallDuration, toBW(totalBytes, overallDuration));

    // free memory buffers on the GPU
    hipFree(device_offsets);
    hipFree(device_neighbours);
    hipFree(device_levels);
}

void
printCudaInfo() {

    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
