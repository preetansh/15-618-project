#include <stdio.h>

#include <hip/hip_runtime.h>

#include <driver_functions.h>

__global__ void
simple_kernel(int N, float* x, float* result) {

    // increment each element by 1
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < N)
       result[index] = x[index] + 1;
}

void simple_gpu_test_function() {
	int N = 10000;
	int totalBytes = sizeof(float) * N;

	// initialize array
	float* x = (float *) malloc (totalBytes);
	float* result = (float *) malloc (totalBytes);
	for (int i=0; i<N; i++) {
		x[i] = i * 1.0;
	}

    // compute number of blocks and threads per block
    const int threadsPerBlock = 512;
    const int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    float* device_x;
    float* device_result;

    // allocate device memory buffers on the GPU using cudaMalloc
    hipMalloc((void **) &device_x, N * sizeof(float));
    hipMalloc((void **) &device_result, N * sizeof(float));

    hipMemcpy(device_x, x, N * sizeof(float), hipMemcpyHostToDevice);

    // run kernel
    simple_kernel<<<blocks, threadsPerBlock>>>(N, device_x, device_result);
    hipDeviceSynchronize();

    // from GPU using cudaMemcpy
    hipMemcpy(result, device_result, N * sizeof(float), hipMemcpyDeviceToHost);

    for (int i=0; i<N; i++) {
    	if (result[i] != (i*1.0)+1) {
    		printf("Incorrect result at %d %f\n", i, result[i]);
    	}
    }

    hipFree(device_x);
    hipFree(device_result);
    free(x);
    free(result);
}